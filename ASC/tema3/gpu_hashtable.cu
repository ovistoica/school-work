#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include <cmath>

#include "gpu_hashtable.hpp"
#include "random_generator.h" // RELIABLE RANDOM GENERATOR

#define MAX_ATTEMPTS 1000
#define MAX_INT 0xffffffff
#define JUMP 41
#define NUM_THREADS 64


const Entry EMPTY = (Entry)0xffffffff << 32;

__global__ void fill_empty(Entry *table, unsigned int table_size) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < table_size) {
		table[id] = EMPTY;
	}
}


__device__ unsigned long jump(int key) {
	return 1 + (key % JUMP);
}

__device__ unsigned long hash_func(int key, unsigned int a, unsigned int b) {
	 return ((a ^ key) + b) % MAX_INT;
}

__device__ int get_key(Entry e) {
	int key = e >> 32;
	return key;
}

__device__ int get_value(Entry e) {
	int val = e - EMPTY;
	return val;
}

__global__ void check_new_items(int *keys, int numKeys, unsigned int table_size,
			      unsigned long a, unsigned long b, Entry *table,
			      unsigned int *new_elems) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < numKeys) {
		int key = keys[id];

		 Entry entry = ((Entry) key << 32);

		/* Computer hash for the key */
		unsigned long index = hash_func(key, a, b);
		index %= table_size;

		/* Compute the jump for this key */
		unsigned long hash_jump = jump(key);
		Entry old_entry;
		/* Linear probing for maximum of MAX_ATTEMPTS or table size */
		for (unsigned long attempt = 1; attempt <= MAX_ATTEMPTS || attempt <=
		     table_size; ++attempt) {

			if (get_key(table[index]) == key) {
				atomicDec(new_elems, 1);
				return;
			}

			/* else jump to the next index */
			index += attempt * hash_jump;
			if (index >= table_size) index = 0;
		}
	}
}


/* INSERT WITH THREADS
 */
__global__ void kernel_insert(int *keys, int *values, int numKeys, unsigned int table_size,
			      unsigned long a, unsigned long b, Entry *table,
			      unsigned int *num_elements) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < numKeys) {
		int key = keys[id];
		int value = values[id];

		 Entry entry = ((Entry) key << 32) + value;

		/* Computer hash for the key */
		unsigned long index = hash_func(key, a, b);
		index %= table_size;

		/* Compute the jump for this key */
		unsigned long hash_jump = jump(key);
		Entry old_entry;
		/* Linear probing for maximum of MAX_ATTEMPTS or table size */
		for (unsigned long attempt = 1; attempt <= MAX_ATTEMPTS || attempt <=
		     table_size; ++attempt) {

			old_entry = atomicCAS(table + index, EMPTY, entry);

			/* If the table slot was empty then finish */
			if (old_entry == EMPTY) {
				atomicAdd(num_elements, 1);
				return;
			} else if (get_key(table[index]) == key) {
				atomicExch(table + index, entry);
				return;
			}

			/* else jump to the next index */
			index += attempt * hash_jump;
			if (index >= table_size) index = 0;
		}
	}
}

__global__ void rehash(Entry *old_table, unsigned int old_size, Entry *new_table,
		       unsigned int new_size, unsigned long a, unsigned long b) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < old_size) {

		/* If no entry in the old table return */
		if (old_table[id] == EMPTY) return;

		/* Compute key and value from old table */
		int key = old_table[id] >> 32;
		int value = old_table[id] - EMPTY;

		 Entry entry = ((Entry) key << 32) + value;

		/* Computer hash for the key */
		unsigned long index = hash_func(key, a, b);
		index %= new_size;

		/* Compute the jump for this key */
		unsigned long hash_jump = jump(key);
		Entry old_entry;
		/* Linear probing for maximum of MAX_ATTEMPTS or table size */
		for (unsigned long attempt = 1; attempt <= MAX_ATTEMPTS || attempt <=
		     new_size; ++attempt) {
			old_entry = atomicCAS(new_table + index, EMPTY, entry);

			/* If the table slot was empty then finish */
			if (old_entry == EMPTY) return;
			/* else jump to the next index */
			index += attempt * hash_jump;
			if (index >= new_size) index = 0;
		}
	}

}

__global__ void kernel_get(int *keys, int *values, int numKeys, Entry *table, unsigned int table_size,
			   unsigned long a, unsigned long b) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < numKeys) {
		int key = keys[id];
		unsigned long index = hash_func(key, a, b);
		index %= table_size;
		unsigned long hash_jump = jump(key);
		/* Linear probing for maximum of MAX_ATTEMPTS or table size */
		for (unsigned long attempt = 1; attempt <= MAX_ATTEMPTS || attempt <=
		     table_size; ++attempt) {
			/* Retrieve value if the key is valid */
			if (get_key(table[index]) == key) {
				values[id] = get_value(table[index]);
				return;
			}

			index += attempt * hash_jump;
			if (index >= table_size) index = 0;
		}

	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
	this->size = size;
	init_genrand(size);

	/* Generate random numbers for hash function */
	this->a = genrand_int32();
	this->b = genrand_int32();

	/* Number of elements is accessed both from CPU and GPU */
	hipMalloc(&this->num_elements, 1 * sizeof(unsigned int));
	hipMemset(this->num_elements, 0, sizeof(unsigned int));

	/* Allocate hash table and insert empty entries */
	hipMalloc((void **)&(this->table), size * sizeof(uint64_t));
	int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;
	fill_empty<<<NUM_BLOCKS,NUM_THREADS>>>(this->table, size);
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(this->table);
	hipFree(this->num_elements);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	Entry *new_table;
	int new_size;

	new_size = ceil(numBucketsReshape * 1.25);
	hipMalloc((void**)&(new_table), new_size * sizeof(Entry));

	int blocks = (new_size + NUM_THREADS - 1) / NUM_THREADS;

	/* Fill new table with empty entries */
	fill_empty<<<blocks,NUM_THREADS>>>(new_table, new_size);

	int NUM_BLOCKS = (this->size + NUM_THREADS - 1) / NUM_THREADS;

	/* Rehashing old entries into the new table */
	rehash<<<NUM_BLOCKS, NUM_THREADS>>>(this->table, this->size, new_table,
					    new_size, this->a, this->b);

	cout << "Rehash Func " << "Reserve arg: " << numBucketsReshape << endl;
	this->printTable();

	hipFree(this->table);
	this->table = new_table;
	this->size = new_size;

	cout << "New table " << endl;
	this->printTable();
	cout << "\n\n";

}

void GpuHashTable::printTable() {
	Entry *my_table;
	int *num_ele;
	num_ele = (int *)malloc(sizeof(int));
	my_table = (Entry *)malloc(sizeof(Entry) * this->size);
	hipMemcpy(num_ele, this->num_elements, sizeof(int),
		   hipMemcpyDeviceToHost);
	hipMemcpy(my_table, this->table, this->size * sizeof(Entry),
		   hipMemcpyDeviceToHost);
	cout << "Size: " << this->size << " Table: \n";
	for(int i = 0; i < this->size; ++i) {
		cout << my_table[i] << " ";
	}
	cout << endl;
	cout << "Num elements: " << *num_ele << endl;
	free(my_table);
	free(num_ele);
}




/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
	int *num_ele;
	num_ele = (int *)malloc(sizeof(int));
	hipMemcpy(num_ele, this->num_elements, sizeof(int),
		   hipMemcpyDeviceToHost);
	// float load = this->loadFactor();
	// if ((numKeys > this->size || numKeys > this->size - *(num_ele)) && load
	//     > 0.5) {
	// 	this->reshape(numKeys);
	// }
	if (*num_ele + numKeys > this->size * 0.7) {
		this->reshape(*num_ele + numKeys);
	}

	int *device_keys, *device_values;
	hipMalloc((void**)&device_keys, numKeys * sizeof(int));
	hipMalloc((void**)&device_values, numKeys * sizeof(int));
	hipMemcpy(device_keys, keys, numKeys * sizeof(int),
		   hipMemcpyHostToDevice);
	hipMemcpy(device_values, values, numKeys * sizeof(int),
		   hipMemcpyHostToDevice);

	int blocks = (numKeys + NUM_THREADS - 1) / NUM_THREADS;
	kernel_insert<<<blocks, NUM_THREADS>>>(device_keys, device_values, numKeys,
					       this->size, this->a, this->b,
					       this->table, this->num_elements);
	cout << "InsertBatch " << "Num keys: " << numKeys << endl;
	this->printTable();
	cout << "\n\n";

	hipFree(device_keys);
	hipFree(device_values);
	free(num_ele);
	return true;
}


/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
	int *device_keys, *device_values, *values;

	hipMalloc((void**)&device_keys, numKeys * sizeof(int));
	hipMalloc((void**)&device_values, numKeys * sizeof(int));
	values = (int *)malloc(numKeys * sizeof(int));
	hipMemcpy(device_keys, keys, numKeys * sizeof(int),
		   hipMemcpyHostToDevice);

	/* Calculate proper block number and launch get kernel */
	int blocks = (numKeys + NUM_THREADS - 1) / NUM_THREADS;
	kernel_get<<<blocks, NUM_THREADS>>>(device_keys, device_values, numKeys,
					    this->table, this->size, this->a,
					    this->b);

	hipMemcpy(values, device_values, numKeys * sizeof(int),
		   hipMemcpyDeviceToHost);
	hipFree(device_keys);
	hipFree(device_values);
	return values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	int *num_ele;
	num_ele = (int *)malloc(sizeof(int));
	hipMemcpy(num_ele, this->num_elements, sizeof(int),
		   hipMemcpyDeviceToHost);

	cout << "Load factor " << "num_ele " << *num_ele << endl;
	float load_fact = *num_ele / (this->size * 1.0f); // no larger than 1.0f = 100%

	cout << "load: " << load_fact << endl;
	cout << "\n\n";
	free(num_ele);

	return load_fact;
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
